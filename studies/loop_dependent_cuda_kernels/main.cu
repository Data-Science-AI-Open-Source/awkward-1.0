#include <iostream>
#include <exception>
#include <sstream>
#include "standard_parallel_algorithms.cuh"

int main() {
	int h_arr[] = {1, 1, 1, 0, 1, 0, 1};

	int* d_in; 
	int* d_out; 
	int* res_dest_final;
	int* res_dest_temp;
	int* d_temp;

	hipError_t err;

	err = hipMalloc((void**)&res_dest_final, sizeof(h_arr));

	if(err != hipError_t::hipSuccess) {
			std::stringstream out;
			out <<  "CUDA ERROR: " <<  hipGetErrorString(err) << "\n";
			throw std::runtime_error(out.str());
	}

	err = hipMalloc((void**)&res_dest_temp, sizeof(h_arr));

	if(err != hipError_t::hipSuccess) {
			std::stringstream out;
			out <<  "CUDA ERROR: " <<  hipGetErrorString(err) << "\n";
			throw std::runtime_error(out.str());
	}

	err = hipMalloc((void**)&d_temp, sizeof(h_arr));

	if(err != hipError_t::hipSuccess) {
			std::stringstream out;
			out <<  "CUDA ERROR: " <<  hipGetErrorString(err) << "\n";
			throw std::runtime_error(out.str());
	}

	err = hipMalloc((void**)&d_in, sizeof(h_arr));

	if(err != hipError_t::hipSuccess) {
			std::stringstream out;
			out <<  "CUDA ERROR: " <<  hipGetErrorString(err) << "\n";
			throw std::runtime_error(out.str());
	}

	err = hipMemcpy(d_in, h_arr, sizeof(h_arr), hipMemcpyHostToDevice);

	if(err != hipError_t::hipSuccess) {
			std::stringstream out;
			out << "CUDA ERROR: " <<  hipGetErrorString(err) << "\n";
			throw std::runtime_error(out.str());
	}
	
	err = hipMemcpy(d_temp, d_in, sizeof(h_arr), hipMemcpyDeviceToDevice);

	if(err != hipError_t::hipSuccess) {
			std::stringstream out;
			out << "CUDA ERROR: " <<  hipGetErrorString(err) << "\n";
			throw std::runtime_error(out.str());
	}
	

	err = hipMalloc((void**)&d_out, sizeof(h_arr));

	if(err != hipError_t::hipSuccess) {
			std::stringstream out;
			out <<  "CUDA ERROR: " <<  hipGetErrorString(err) << "\n";
			throw std::runtime_error(out.str());
	}


	exclusive_scan(&res_dest_temp, d_in, sizeof(h_arr) / sizeof(int));

	scatter<<<1, sizeof(h_arr) / sizeof(int)>>>(res_dest_temp, d_temp, res_dest_final);

	
	err = hipMemcpy(h_arr, res_dest_final, sizeof(h_arr), hipMemcpyDeviceToHost);

	if(err != hipError_t::hipSuccess) {
			std::stringstream out;
			out << "CUDA ERROR: " <<  hipGetErrorString(err) << "\n";
			throw std::runtime_error(out.str());
	}

	for(int i = 0; i < sizeof(h_arr) / sizeof(int); i++) {
			std::cout << h_arr[i] << " ";
	}
	std::cout << std::endl;
}
